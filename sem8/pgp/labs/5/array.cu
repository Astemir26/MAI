#include "hip/hip_runtime.h"
#include "array.h"

void arrayRead(Array* arr)
{
	freopen(NULL, "rb", stdin);
	fread(&arr->size, sizeof(arr->size), 1, stdin);

	arr->data = (Byte*)malloc(sizeof(Byte) * arr->size);

	fread(arr->data, sizeof(Byte), arr->size, stdin);
	fclose(stdin);
}

void arrayWrite(Array* arr)
{
	freopen(NULL, "wb", stdout);
	fwrite(arr->data, sizeof(Byte), arr->size, stdout);
	free(arr->data);
	fclose(stdout);
}

void arraySort(Array* arr)
{
	Byte* dArr;
	int* dHist;
	int* dPrefix;

	int histSize = sizeof(int) * 256;

	ERR(hipMalloc(&dArr, arr->size));
	ERR(hipMalloc(&dHist, histSize));
	ERR(hipMalloc(&dPrefix, histSize));
	ERR(hipMemcpy(dArr, arr->data, arr->size, hipMemcpyHostToDevice));
	ERR(hipMemset(dHist, 0, histSize));

	histogramKernel<<<32, 32>>>(dArr, dHist, arr->size);
	scanKernel<<<1, 256>>>(dHist, dPrefix);
	arrangementKernel<<<8, 32>>>(dArr, dHist, dPrefix);

	ERR(hipMemcpy(arr->data, dArr, arr->size, hipMemcpyDeviceToHost));
	ERR(hipFree(dArr));
	ERR(hipFree(dHist));
	ERR(hipFree(dPrefix));
}

__device__ int arrayConflictFree(int index)
{
	return index + (index >> 5);
}

__global__ void histogramKernel(Byte* arr, int* hist, int arrCount)
{
	__shared__ int temp[256];

	int tIdLocal = threadIdx.x;
	int tIdGlobal = blockDim.x * blockIdx.x + tIdLocal;
	int offsetX = gridDim.x * blockDim.x;

	for (int i = tIdGlobal; i < arrCount; i += offsetX)
		atomicAdd(&temp[arr[i]], 1);

	__syncthreads();

	for (int i = tIdLocal; i < 256; i += blockDim.x)
		atomicAdd(&hist[i], temp[i]);
}

__global__ void scanKernel(int* hist, int* prefix)
{
	__shared__ int temp[256 + 6];

	int tId = threadIdx.x;
	int offset = 1;

	temp[arrayConflictFree(tId)] = hist[tId];

	for (int d = 256 >> 1; d > 0; d >>= 1)
	{
		__syncthreads();

		if (tId < d)
		{
			int index1 = arrayConflictFree(offset * (tId * 2 + 1) - 1);
			int index2 = arrayConflictFree(offset * (tId * 2 + 2) - 1);

			temp[index2] += temp[index1];
		}

		offset <<= 1;
	}

	if (tId == 255)
		temp[arrayConflictFree(255)] = 0;

	for (int d = 1; d < 256; d <<= 1)
	{
		offset >>= 1;

		__syncthreads();

		if (tId < d)
		{
			int index1 = arrayConflictFree(offset * (tId * 2 + 1) - 1);
			int index2 = arrayConflictFree(offset * (tId * 2 + 2) - 1);
			int t = temp[index1];

			temp[index1] = temp[index2];
			temp[index2] += t;
		}
	}

	__syncthreads(); 
	
	if (tId < 255)
		prefix[tId] = temp[arrayConflictFree(tId + 1)];
	else
		prefix[tId] = temp[arrayConflictFree(tId)] + hist[255];
}

__global__ void arrangementKernel(Byte* arr, int* hist, int* prefix)
{
	__shared__ int temp[256 * 2];

	int tId = blockDim.x * blockIdx.x + threadIdx.x;

	temp[tId] = hist[tId];
	temp[tId + 256] = prefix[tId];	

	while (temp[tId] > 0)
	{
		arr[temp[tId + 256] - 1] = (Byte)tId;
		--temp[tId + 256];
		--temp[tId];
	}
}
