#include "hip/hip_runtime.h"
#include "array.h"

void arrayRead(Array* arr)
{
	freopen(NULL, "rb", stdin);
	fread(&arr->size, sizeof(arr->size), 1, stdin);

	arr->data = (Byte*)malloc(sizeof(Byte) * arr->size);

	fread(arr->data, sizeof(Byte), arr->size, stdin);
	fclose(stdin);
}

void arrayWrite(Array* arr)
{
	freopen(NULL, "wb", stdout);
	fwrite(arr->data, sizeof(Byte), arr->size, stdout);
	free(arr->data);
	fclose(stdout);
}

void arraySort(Array* arr)
{
	Byte* dArrSrc;
	Byte* dArrRes;
	int* dHist;
	int* dPrefix;

	int histSize = sizeof(int) * BLOCK_SIZE;

	ERR(hipMalloc(&dArrSrc, arr->size));
	ERR(hipMalloc(&dArrRes, arr->size));
	ERR(hipMalloc(&dHist, histSize));
	ERR(hipMalloc(&dPrefix, histSize));
	ERR(hipMemcpy(dArrSrc, arr->data, arr->size, hipMemcpyHostToDevice));
	ERR(hipMemset(dHist, 0, histSize));

	histogramKernel<<<32, 32>>>(dArrSrc, dHist, arr->size);
	scanKernel<<<1, BLOCK_SIZE>>>(dHist, dPrefix);
	arrangementKernel<<<32, 32>>>(dArrSrc, dArrRes, dPrefix, arr->size);

	ERR(hipGetLastError());
	ERR(hipMemcpy(arr->data, dArrRes, arr->size, hipMemcpyDeviceToHost));
	ERR(hipFree(dArrSrc));
	ERR(hipFree(dArrRes));
	ERR(hipFree(dHist));
	ERR(hipFree(dPrefix));
}

__device__ int conflictFree(int index)
{
	return index + (index >> LOG2_BANKS);
}

__global__ void histogramKernel(Byte* arr, int* hist, int arrCount)
{
	__shared__ int temp[BLOCK_SIZE];

	int tIdLocal = threadIdx.x;
	int tIdGlobal = blockDim.x * blockIdx.x + tIdLocal;
	int offsetX = gridDim.x * blockDim.x;

	for (int i = tIdGlobal; i < arrCount; i += offsetX)
		atomicAdd(temp + arr[i], 1);

	__syncthreads();

	for (int i = tIdLocal; i < BLOCK_SIZE; i += blockDim.x)
		atomicAdd(hist + i, temp[i]);
}

__global__ void scanKernel(int* hist, int* prefix)
{
	__shared__ int temp[BLOCK_SIZE + BLOCK_SIZE / 32];

	int tId = threadIdx.x;
	int offset = 1;

	temp[conflictFree(tId)] = hist[tId];

	for (int d = BLOCK_SIZE >> 1; d > 0; d >>= 1)
	{
		__syncthreads();

		if (tId < d)
		{
			int index1 = conflictFree(offset * (tId * 2 + 1) - 1);
			int index2 = conflictFree(offset * (tId * 2 + 2) - 1);

			temp[index2] += temp[index1];
		}

		offset <<= 1;
	}

	if (tId == BLOCK_SIZE - 1)
		temp[conflictFree(BLOCK_SIZE - 1)] = 0;

	for (int d = 1; d < BLOCK_SIZE; d <<= 1)
	{
		offset >>= 1;

		__syncthreads();

		if (tId < d)
		{
			int index1 = conflictFree(offset * (tId * 2 + 1) - 1);
			int index2 = conflictFree(offset * (tId * 2 + 2) - 1);
			int t = temp[index1];

			temp[index1] = temp[index2];
			temp[index2] += t;
		}
	}

	__syncthreads(); 
	
	if (tId < BLOCK_SIZE - 1)
		prefix[tId] = temp[conflictFree(tId + 1)];
	else
		prefix[tId] = temp[conflictFree(tId)] + hist[BLOCK_SIZE - 1];
}

__global__ void arrangementKernel(Byte* arrSrc, Byte* arrRes, int* prefix, int arrCount)
{
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	int offsetX = gridDim.x * blockDim.x;

	for (int i = tId; i < arrCount; i += offsetX)
	{
		int pos = atomicSub(prefix + arrSrc[i], 1) - 1;

		arrRes[pos] = arrSrc[i];
	}
}
