
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define ERR(call) \
{ \
	hipError_t err = call; \
	\
	if (err != hipSuccess) \
	{ \
		fprintf(stderr, "ERROR: CUDA failed in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
		exit(0); \
	} \
}

__global__ void kernel(double* dA, double* dB, double* dC, int n);

int main(void)
{
	int n;
	int size;
	double* hA;
	double* hB;
	double* hC;
	double* dA;
	double* dB;
	double* dC;

	scanf("%d", &n);

	size = sizeof(double) * n;
	hA = (double*)malloc(size);
	hB = (double*)malloc(size);
	hC = (double*)malloc(size);

	for (int i = 0; i < n; ++i)
		scanf("%lf", &hA[i]);

	for (int i = 0; i < n; ++i)
		scanf("%lf", &hB[i]);

	ERR(hipMalloc(&dA, size));
	ERR(hipMalloc(&dB, size));
	ERR(hipMalloc(&dC, size));
	ERR(hipMemcpy(dA, hA, size, hipMemcpyHostToDevice));
	ERR(hipMemcpy(dB, hB, size, hipMemcpyHostToDevice));

	kernel<<<256, 256>>>(dA, dB, dC, n);

	ERR(hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost));

	for (int i = 0; i < n; ++i)
		printf("%.10e ", hC[i]);

	printf("\n");

	ERR(hipFree(dC));
	ERR(hipFree(dB));
	ERR(hipFree(dA));
	free(hC);
	free(hB);
	free(hA);

	return 0;
}

__global__ void kernel(double* dA, double* dB, double* dC, int n)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = gridDim.x * blockDim.x;

	while (idx < n)
	{
		dC[idx] = dA[idx] * dB[idx];
		idx += offset;
	}
}
