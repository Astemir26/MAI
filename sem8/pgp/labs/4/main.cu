#include "gauss.h"

int main(void)
{
	Gauss gauss;
/*
	hipEvent_t start;
	hipEvent_t stop;
	float delta;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
*/
	gaussCreate(&gauss);
	gaussSolve(&gauss);	
	gaussPrintResult(&gauss);
	gaussDelete(&gauss);
/*
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&delta, start, stop);

	printf("Time: %f\n", delta);
*/
	return 0;
}
