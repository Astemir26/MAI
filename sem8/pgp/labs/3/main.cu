#include "classifier.h"

int main(void)
{
	char inputFileName[256];
	char outputFileName[256];
	Classifier classifier;
	Image image;
	Pixel* dPixels;

	scanf("%s", inputFileName);
	scanf("%s", outputFileName);

	imageReadFromFile(&image, inputFileName);

	classifierCreate(&classifier);
	classifierCalc(&classifier, &image);
	classifierCopyToConstant(&classifier);
	classifierDelete(&classifier);

	ERR(hipMalloc(&dPixels, imageSize(&image)));
	ERR(hipMemcpy(dPixels, image.pixels, imageSize(&image), hipMemcpyHostToDevice));

	dim3 gridSize(32, 32);
	dim3 blockSize(32, 32);

	classifierMahalanobisKernel<<<gridSize, blockSize>>>(dPixels, image.width, image.height);

	ERR(hipMemcpy(image.pixels, dPixels, imageSize(&image), hipMemcpyDeviceToHost));
	ERR(hipFree(dPixels));

	imageWriteToFile(&image, outputFileName);
	imageDelete(&image);

	return 0;
}
